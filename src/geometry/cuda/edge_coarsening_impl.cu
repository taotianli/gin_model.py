#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file geometry/cuda/edge_coarsening_impl.cu
 * \brief Edge coarsening CUDA implementation
 */
#include <dgl/array.h>
#include <dgl/random.h>
#include <dmlc/thread_local.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdint>
#include "../geometry_op.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../../array/cuda/utils.h"

#define BLOCKS(N, T) (N + T - 1) / T

namespace dgl {
namespace geometry {
namespace impl {

constexpr float BLUE_P = 0.53406;
constexpr int BLUE = -1;
constexpr int RED = -2;
constexpr int EMPTY_IDX = -1;

__device__ bool done_d;
__global__ void init_done_kernel() { done_d = true; }

__global__ void generate_uniform_kernel(float *ret_values, size_t num, uint64_t seed) {
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < num) {
    hiprandState state;
    hiprand_init(seed, id, 0, &state);
    ret_values[id] = hiprand_uniform(&state);
  }
}

template <typename IdType>
__global__ void colorize_kernel(const float *prop, int64_t num_elem, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] < 0) {  // if unmatched
      result[idx] = (prop[idx] > BLUE_P) ? RED : BLUE;
      done_d = false;
    }
  }
}

template <typename FloatType, typename IdType>
__global__ void weighted_propose_kernel(const IdType *indptr, const IdType *indices,
                                        const FloatType *weights, int64_t num_elem,
                                        IdType *proposal, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] != BLUE) return;

    bool has_unmatched_neighbor = false;
    FloatType weight_max = 0.;
    IdType v_max = EMPTY_IDX;

    for (IdType i = indptr[idx]; i < indptr[idx + 1]; ++i) {
      auto v = indices[i];

      if (result[v] < 0)
        has_unmatched_neighbor = true;
      if (result[v] == RED && weights[i] >= weight_max) {
        v_max = v;
        weight_max = weights[i];
      }
    }

    proposal[idx] = v_max;
    if (!has_unmatched_neighbor)
      result[idx] = idx;
  }
}

template <typename FloatType, typename IdType>
__global__ void weighted_respond_kernel(const IdType *indptr, const IdType *indices,
                                        const FloatType *weights, int64_t num_elem,
                                        IdType *proposal, IdType *result) {
  const IdType idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elem) {
    if (result[idx] != RED) return;

    bool has_unmatched_neighbors = false;
    IdType v_max = -1;
    FloatType weight_max = 0.;

    for (IdType i = indptr[idx]; i < indptr[idx + 1]; ++i) {
      auto v = indices[i];

      if (result[v] < 0) {
        has_unmatched_neighbors = true;
      }
      if (result[v] == BLUE
          && proposal[v] == idx
          && weights[i] >= weight_max) {
        v_max = v;
        weight_max = weights[i];
      }
    }
    if (v_max >= 0) {
      result[v_max] = min(idx, v_max);
      result[idx] = min(idx, v_max);
    }

    if (!has_unmatched_neighbors)
      result[idx] = idx;
  }
}

/*! \brief The colorize procedure. This procedure randomly marks unmarked
 * nodes with BLUE(-1) and RED(-2) and checks whether the node matching
 * process has finished.
 */
template<typename IdType>
bool Colorize(IdType * result_data, int64_t num_nodes, float * const prop) {
  // initial done signal
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  CUDA_KERNEL_CALL(init_done_kernel, 1, 1, 0, thr_entry->stream);

  // generate color prop for each node
  uint64_t seed = dgl::RandomEngine::ThreadLocal()->RandInt(UINT64_MAX);
  auto num_threads = cuda::FindNumThreads(num_nodes);
  auto num_blocks = cuda::FindNumBlocks<'x'>(BLOCKS(num_nodes, num_threads));
  CUDA_KERNEL_CALL(generate_uniform_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                   prop, num_nodes, seed);

  // call kernel
  CUDA_KERNEL_CALL(colorize_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                   prop, num_nodes, result_data);
  bool done_h = false;
  CUDA_CALL(hipMemcpyFromSymbol(&done_h, HIP_SYMBOL(done_d), sizeof(done_h), 0, hipMemcpyDeviceToHost));
  return done_h;
}

/*! \brief Weighted neighbor matching procedure (GPU version).
 * This implementation is from `A GPU Algorithm for Greedy Graph Matching
 * <http://www.staff.science.uu.nl/~bisse101/Articles/match12.pdf>`__
 * 
 * This algorithm has three parts: colorize, propose and respond.
 * In colorize procedure, each unmarked node will be marked as BLUE or
 * RED randomly. If all nodes are marked, finish and return.
 * In propose procedure, each BLUE node will propose to the RED
 * neighbor with the largest weight (or randomly choose one if without weight).
 * If all its neighbors are marked, mark this node with its id.
 * In respond procedure, each RED node will respond to the BLUE neighbor
 * that has proposed to it and has the largest weight. If all neighbors
 * are marked, mark this node with its id. Else match this (BLUE, RED) node
 * pair and mark them with the smaller id between them.
 */
template <DLDeviceType XPU, typename FloatType, typename IdType>
void WeightedNeighborMatching(const aten::CSRMatrix &csr, const NDArray weight, IdArray result) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const auto& ctx = result->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  device->SetDevice(ctx);

  // create proposal tensor
  const int64_t num_nodes = result->shape[0];
  IdArray proposal = aten::Full(-1, num_nodes, sizeof(IdType) * 8, ctx);

  // get data ptrs
  IdType *indptr_data = static_cast<IdType*>(csr.indptr->data);
  IdType *indices_data = static_cast<IdType*>(csr.indices->data);
  IdType *result_data = static_cast<IdType*>(result->data);
  IdType *proposal_data = static_cast<IdType*>(proposal->data);
  FloatType *weight_data = static_cast<FloatType*>(weight->data);

  // allocate workspace for prop used in Colorize()
  float *prop = static_cast<float*>(
      device->AllocWorkspace(ctx, num_nodes * sizeof(float)));

  auto num_threads = cuda::FindNumThreads(num_nodes);
  auto num_blocks = cuda::FindNumBlocks<'x'>(BLOCKS(num_nodes, num_threads));
  while (!Colorize<IdType>(result_data, num_nodes, prop)) {
    CUDA_KERNEL_CALL(weighted_propose_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                     indptr_data, indices_data, weight_data, num_nodes, proposal_data, result_data);
    CUDA_KERNEL_CALL(weighted_respond_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                     indptr_data, indices_data, weight_data, num_nodes, proposal_data, result_data);
  }
  device->FreeWorkspace(ctx, prop);
}
template void WeightedNeighborMatching<kDLGPU, float, int32_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, float, int64_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, double, int32_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);
template void WeightedNeighborMatching<kDLGPU, double, int64_t>(
  const aten::CSRMatrix &csr, const NDArray weight, IdArray result);

/*! \brief Unweighted neighbor matching procedure (GPU version).
 * Instead of directly sample neighbors, we assign each neighbor
 * with a random weight. We use random weight for 2 reasons:
 *  1. Random sample for each node in GPU is expensive. Although
 *     we can perform a global group-wise (neighborhood of each
 *     node as a group) random permutation as in CPU version,
 *     it still cost too much compared to directly using random weights.
 *  2. Graph is sparse, thus neighborhood of each node is small,
 *     which is suitable for GPU implementation.
 */
template <DLDeviceType XPU, typename IdType>
void NeighborMatching(const aten::CSRMatrix &csr, IdArray result) {
  const int64_t num_edges = csr.indices->shape[0];
  const auto& ctx = result->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  device->SetDevice(ctx);

  // generate random weights
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  NDArray weight = NDArray::Empty(
    {num_edges}, DLDataType{kDLFloat, sizeof(float) * 8, 1}, ctx);
  float *weight_data = static_cast<float*>(weight->data);
  uint64_t seed = dgl::RandomEngine::ThreadLocal()->RandInt(UINT64_MAX);
  auto num_threads = cuda::FindNumThreads(num_edges);
  auto num_blocks = cuda::FindNumBlocks<'x'>(BLOCKS(num_edges, num_threads));
  CUDA_KERNEL_CALL(generate_uniform_kernel, num_blocks, num_threads, 0, thr_entry->stream,
                   weight_data, num_edges, seed);

  WeightedNeighborMatching<XPU, float, IdType>(csr, weight, result);
}
template void NeighborMatching<kDLGPU, int32_t>(const aten::CSRMatrix &csr, IdArray result);
template void NeighborMatching<kDLGPU, int64_t>(const aten::CSRMatrix &csr, IdArray result);

}  // namespace impl
}  // namespace geometry
}  // namespace dgl

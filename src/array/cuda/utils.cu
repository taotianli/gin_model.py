/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/utils.cu
 * \brief Utilities for CUDA kernels.
 */

#include "./utils.h"
#include "./dgl_cub.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {
namespace cuda {

bool AllTrue(int8_t* flags, int64_t length, const DLContext& ctx) {
  auto device = runtime::DeviceAPI::Get(ctx);
  int8_t* rst = static_cast<int8_t*>(device->AllocWorkspace(ctx, 1));
  // Call CUB's reduction
  size_t workspace_size = 0;
  CUDA_CALL(hipcub::DeviceReduce::Min(nullptr, workspace_size, flags, rst, length));
  void* workspace = device->AllocWorkspace(ctx, workspace_size);
  CUDA_CALL(hipcub::DeviceReduce::Min(workspace, workspace_size, flags, rst, length));
  int8_t cpu_rst = GetCUDAScalar(device, ctx, rst, static_cast<hipStream_t>(0));
  device->FreeWorkspace(ctx, workspace);
  device->FreeWorkspace(ctx, rst);
  return cpu_rst == 1;
}

}  // namespace cuda
}  // namespace dgl

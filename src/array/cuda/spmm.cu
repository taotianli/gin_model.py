#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/spmm.cu
 * \brief SPMM C APIs and definitions.
 */
#include <dgl/array.h>
#include "./spmm.cuh"
#include "./ge_spmm.cuh"
#include "./functor.cuh"
#include "../../runtime/cuda/cuda_common.h"

namespace dgl {

using namespace cuda;

namespace aten {

/*!
 * \brief Determine whether cusparse SpMM function is applicable.
 */
template <int bits, typename IdType>
inline bool cusparse_available(bool more_nnz_than_matrix_size) {
#if CUDART_VERSION < 11000
  if (std::is_same<IdType, int>::value)
    if (bits > 16)
      return true;
  return false;
#else
  if (bits == 16)
    return false;  // cusparse's SpMM on fp16 is slow, temporally disabled.
  // If the CSR matrix has more NNZ than matrix size, we should not use cuSPARSE 11.1.
  return !more_nnz_than_matrix_size;
#endif
}

/*!
 * \brief CUDA implementation of g-SpMM on Csr format.
 * \note use cusparse if the reduce operator is `sum` and there is
 *       no broadcast, use dgl's kernel in other cases.
 */
template <int XPU, typename IdType, int bits>
void SpMMCsr(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const CSRMatrix& csr,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  bool is_scalar_efeat = efeat.NumElements() == csr.indices->shape[0];
  bool use_efeat = op != "copy_lhs";

  if (reduce == "sum") {
    bool more_nnz = (csr.indices->shape[0] > csr.num_rows * csr.num_cols);
    if (op == "copy_lhs" && cusparse_available<bits, IdType>(more_nnz)) {
      // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      SWITCH_BITS(bits, DType, {
        CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            nullptr,
            static_cast<DType*>(out->data),
            x_length);
      });
    } else if (op == "mul" && is_scalar_efeat && cusparse_available<bits, IdType>(more_nnz)) {
      // cusparse
      int64_t x_length = 1;
      for (int i = 1; i < ufeat->ndim; ++i)
        x_length *= ufeat->shape[i];
      if (!IsNullArray(csr.data)) {
        SWITCH_BITS(bits, DType, {
          efeat = _IndexSelect<DType, IdType>(efeat, csr.data);
        });
      }
      SWITCH_BITS(bits, DType, {
        CusparseCsrmm2<DType, IdType>(
            ufeat->ctx, csr,
            static_cast<DType*>(ufeat->data),
            static_cast<DType*>(efeat->data),
            static_cast<DType*>(out->data),
            x_length);
      });
    } else {  // general kernel
      SWITCH_BITS(bits, DType, {
        SWITCH_OP(op, Op, {
          cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Sum<IdType, DType> >(
              bcast, csr, ufeat, efeat, out, NullArray(), NullArray());
        });
      });
    }
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Max<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCsr<IdType, DType, Op, cuda::reduce::Min<IdType, DType> >(
            bcast, csr, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}


/*!
 * \brief CUDA implementation of g-SpMM on Coo format.
 */
template <int XPU, typename IdType, int bits>
void SpMMCoo(const std::string& op, const std::string& reduce,
             const BcastOff& bcast,
             const COOMatrix& coo,
             NDArray ufeat,
             NDArray efeat,
             NDArray out,
             std::vector<NDArray> out_aux) {
  if (reduce == "sum") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Sum<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, NullArray(), NullArray());
      });
    });
  } else if (reduce == "max") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Max<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  }  else if (reduce == "min") {
    SWITCH_BITS(bits, DType, {
      SWITCH_OP(op, Op, {
        cuda::SpMMCoo<IdType, DType, Op, cuda::reduce::Min<IdType, DType, true> > (
            bcast, coo, ufeat, efeat, out, out_aux[0], out_aux[1]);
      });
    });
  } else {
    LOG(FATAL) << "Not implemented";
  }
}

template void SpMMCsr<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCsr<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const CSRMatrix& csr,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);


template void SpMMCoo<kDLGPU, int32_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 16>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 32>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int32_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);
template void SpMMCoo<kDLGPU, int64_t, 64>(
    const std::string& op, const std::string& reduce,
    const BcastOff& bcast, const COOMatrix& coo,
    NDArray ufeat, NDArray efeat, NDArray out, std::vector<NDArray> out_aux);


}  // namespace aten
}  // namespace dgl

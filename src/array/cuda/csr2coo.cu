#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/csr2coo.cc
 * \brief CSR2COO
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

template <DLDeviceType XPU, typename IdType>
COOMatrix CSRToCOO(CSRMatrix csr) {
  LOG(FATAL) << "Unreachable codes";
  return {};
}

template <>
COOMatrix CSRToCOO<kDLGPU, int32_t>(CSRMatrix csr) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));

  NDArray indptr = csr.indptr, indices = csr.indices, data = csr.data;
  const int32_t* indptr_ptr = static_cast<int32_t*>(indptr->data);
  NDArray row = aten::NewIdArray(indices->shape[0], indptr->ctx, indptr->dtype.bits);
  int32_t* row_ptr = static_cast<int32_t*>(row->data);

  CUSPARSE_CALL(hipsparseXcsr2coo(
      thr_entry->cusparse_handle,
      indptr_ptr,
      indices->shape[0],
      csr.num_rows,
      row_ptr,
      HIPSPARSE_INDEX_BASE_ZERO));

  return COOMatrix(csr.num_rows, csr.num_cols,
                   row, indices, data,
                   true, csr.sorted);
}

/*!
 * \brief Repeat elements
 * \param val Value to repeat
 * \param repeats Number of repeats for each value
 * \param pos The position of the output buffer to write the value.
 * \param out Output buffer.
 * \param length Number of values
 *
 * For example:
 * val = [3, 0, 1]
 * repeats = [1, 0, 2]
 * pos = [0, 1, 1]  # write to output buffer position 0, 1, 1
 * then,
 * out = [3, 1, 1]
 */
template <typename DType, typename IdType>
__global__ void _RepeatKernel(
    const DType* val, const IdType* pos,
    DType* out, int64_t n_row, int64_t length) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    IdType i = dgl::cuda::_UpperBound(pos, n_row, tx) - 1;
    out[tx] = val[i];
    tx += stride_x;
  }
}

template <>
COOMatrix CSRToCOO<kDLGPU, int64_t>(CSRMatrix csr) {
  const auto& ctx = csr.indptr->ctx;
  const int64_t nnz = csr.indices->shape[0];
  const auto nbits = csr.indptr->dtype.bits;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  IdArray rowids = Range(0, csr.num_rows, nbits, ctx);
  IdArray ret_row = NewIdArray(nnz, ctx, nbits);

  const int nt = 256;
  const int nb = (nnz + nt - 1) / nt;
  CUDA_KERNEL_CALL(_RepeatKernel,
      nb, nt, 0, thr_entry->stream,
      rowids.Ptr<int64_t>(),
      csr.indptr.Ptr<int64_t>(), ret_row.Ptr<int64_t>(),
      csr.num_rows, nnz);

  return COOMatrix(csr.num_rows, csr.num_cols,
                   ret_row, csr.indices, csr.data,
                   true, csr.sorted);
}

template COOMatrix CSRToCOO<kDLGPU, int32_t>(CSRMatrix csr);
template COOMatrix CSRToCOO<kDLGPU, int64_t>(CSRMatrix csr);

template <DLDeviceType XPU, typename IdType>
COOMatrix CSRToCOODataAsOrder(CSRMatrix csr) {
  LOG(FATAL) << "Unreachable codes";
  return {};
}

template <>
COOMatrix CSRToCOODataAsOrder<kDLGPU, int32_t>(CSRMatrix csr) {
  COOMatrix coo = CSRToCOO<kDLGPU, int32_t>(csr);
  if (aten::IsNullArray(coo.data))
    return coo;

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(coo.row->ctx);
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  CUSPARSE_CALL(hipsparseSetStream(thr_entry->cusparse_handle, thr_entry->stream));

  NDArray row = coo.row, col = coo.col, data = coo.data;
  int32_t* row_ptr = static_cast<int32_t*>(row->data);
  int32_t* col_ptr = static_cast<int32_t*>(col->data);
  int32_t* data_ptr = static_cast<int32_t*>(data->data);

  size_t workspace_size = 0;
  CUSPARSE_CALL(hipsparseXcoosort_bufferSizeExt(
      thr_entry->cusparse_handle,
      coo.num_rows, coo.num_cols,
      row->shape[0],
      data_ptr,
      row_ptr,
      &workspace_size));
  void* workspace = device->AllocWorkspace(row->ctx, workspace_size);
  CUSPARSE_CALL(hipsparseXcoosortByRow(
      thr_entry->cusparse_handle,
      coo.num_rows, coo.num_cols,
      row->shape[0],
      data_ptr,
      row_ptr,
      col_ptr,
      workspace));
  device->FreeWorkspace(row->ctx, workspace);

  // The row and column field have already been reordered according
  // to data, thus the data field will be deprecated.
  coo.data = aten::NullArray();
  coo.row_sorted = false;
  coo.col_sorted = false;
  return coo;
}

template <>
COOMatrix CSRToCOODataAsOrder<kDLGPU, int64_t>(CSRMatrix csr) {
  COOMatrix coo = CSRToCOO<kDLGPU, int64_t>(csr);
  if (aten::IsNullArray(coo.data))
    return coo;
  const auto& sorted = Sort(coo.data);

  coo.row = IndexSelect(coo.row, sorted.second);
  coo.col = IndexSelect(coo.col, sorted.second);

  // The row and column field have already been reordered according
  // to data, thus the data field will be deprecated.
  coo.data = aten::NullArray();
  coo.row_sorted = false;
  coo.col_sorted = false;
  return coo;
}

template COOMatrix CSRToCOODataAsOrder<kDLGPU, int32_t>(CSRMatrix csr);
template COOMatrix CSRToCOODataAsOrder<kDLGPU, int64_t>(CSRMatrix csr);

}  // namespace impl
}  // namespace aten
}  // namespace dgl

#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cuda/coo_sort.cc
 * \brief Sort COO index
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "../../c_api_common.h"
#include "./utils.h"

namespace dgl {

using runtime::NDArray;

namespace aten {
namespace impl {

///////////////////////////// COOSort_ /////////////////////////////

/**
* @brief Encode row and column IDs into a single scalar per edge.
*
* @tparam IdType The type to encode as.
* @param row The row (src) IDs per edge.
* @param col The column (dst) IDs per edge.
* @param nnz The number of edges.
* @param col_bits The number of bits used to encode the destination. The row
* information is packed into the remaining bits.
* @param key The encoded edges (output).
*/
template <typename IdType>
__global__ void _COOEncodeEdgesKernel(
    const IdType* const row, const IdType* const col,
    const int64_t nnz, const int col_bits, IdType * const key) {

  int64_t tx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;

  if (tx < nnz) {
    key[tx] = row[tx] << col_bits | col[tx];
  }
}

/**
* @brief Decode row and column IDs from the encoded edges.
*
* @tparam IdType The type the edges are encoded as.
* @param key The encoded edges.
* @param nnz The number of edges.
* @param col_bits The number of bits used to store the column/dst ID.
* @param row The row (src) IDs per edge (output).
* @param col The col (dst) IDs per edge (output).
*/
template <typename IdType>
__global__ void _COODecodeEdgesKernel(
    const IdType* const key, const int64_t nnz, const int col_bits,
    IdType * const row, IdType * const col) {

  int64_t tx = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;

  if (tx < nnz) {
    const IdType k = key[tx];
    row[tx] = k >> col_bits;
    col[tx] = k & ((1 << col_bits) - 1);
  }
}



template<typename T>
int _NumberOfBits(const T& range) {
  if (range <= 1) {
    // ranges of 0 or 1 require no bits to store
    return 0;
  }

  int bits = 1;
  while (bits < sizeof(T)*8 && (1 << bits) < range) {
    ++bits;
  }

  CHECK_EQ((range-1) >> bits, 0);
  CHECK_NE((range-1) >> (bits-1), 0);

  return bits;
}

template <DLDeviceType XPU, typename IdType>
void COOSort_(COOMatrix* coo, bool sort_column) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const int row_bits = _NumberOfBits(coo->num_rows);

  const int64_t nnz = coo->row->shape[0];
  if (sort_column) {
    const int col_bits = _NumberOfBits(coo->num_cols);
    const int num_bits = row_bits + col_bits;

    const int nt = 256;
    const int nb = (nnz+nt-1)/nt;
    CHECK(static_cast<int64_t>(nb)*nt >= nnz);

    IdArray pos = aten::NewIdArray(nnz, coo->row->ctx, coo->row->dtype.bits);

    CUDA_KERNEL_CALL(_COOEncodeEdgesKernel, nb, nt, 0, thr_entry->stream,
        coo->row.Ptr<IdType>(), coo->col.Ptr<IdType>(),
        nnz, col_bits, pos.Ptr<IdType>());

    auto sorted = Sort(pos, num_bits);

    CUDA_KERNEL_CALL(_COODecodeEdgesKernel, nb, nt, 0, thr_entry->stream,
        sorted.first.Ptr<IdType>(), nnz, col_bits,
        coo->row.Ptr<IdType>(), coo->col.Ptr<IdType>());

    if (aten::COOHasData(*coo))
      coo->data = IndexSelect(coo->data, sorted.second);
    else
      coo->data = AsNumBits(sorted.second, coo->row->dtype.bits);
    coo->row_sorted = coo->col_sorted = true;
  } else {
    const int num_bits = row_bits;

    auto sorted = Sort(coo->row, num_bits);

    coo->row = sorted.first;
    coo->col = IndexSelect(coo->col, sorted.second);

    if (aten::COOHasData(*coo))
      coo->data = IndexSelect(coo->data, sorted.second);
    else
      coo->data = AsNumBits(sorted.second, coo->row->dtype.bits);
    coo->row_sorted = true;
  }
}

template void COOSort_<kDLGPU, int32_t>(COOMatrix* coo, bool sort_column);
template void COOSort_<kDLGPU, int64_t>(COOMatrix* coo, bool sort_column);

///////////////////////////// COOIsSorted /////////////////////////////

template <typename IdType>
__global__ void _COOIsSortedKernel(
    const IdType* row, const IdType* col,
    int64_t nnz, int8_t* row_sorted, int8_t* col_sorted) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < nnz) {
    if (tx == 0) {
      row_sorted[0] = 1;
      col_sorted[0] = 1;
    } else {
      row_sorted[tx] = static_cast<int8_t>(row[tx - 1] <= row[tx]);
      col_sorted[tx] = static_cast<int8_t>(
          row[tx - 1] < row[tx] || col[tx - 1] <= col[tx]);
    }
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
std::pair<bool, bool> COOIsSorted(COOMatrix coo) {
  const int64_t nnz = coo.row->shape[0];
  const auto& ctx = coo.row->ctx;
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(ctx);
  // We allocate a workspace of 2*nnz bytes. It wastes a little bit memory but should
  // be fine.
  int8_t* row_flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, nnz));
  int8_t* col_flags = static_cast<int8_t*>(device->AllocWorkspace(ctx, nnz));
  const int nt = cuda::FindNumThreads(nnz);
  const int nb = (nnz + nt - 1) / nt;
  CUDA_KERNEL_CALL(_COOIsSortedKernel, nb, nt, 0, thr_entry->stream,
      coo.row.Ptr<IdType>(), coo.col.Ptr<IdType>(),
      nnz, row_flags, col_flags);

  const bool row_sorted = cuda::AllTrue(row_flags, nnz, ctx);
  const bool col_sorted = row_sorted? cuda::AllTrue(col_flags, nnz, ctx) : false;

  device->FreeWorkspace(ctx, row_flags);
  device->FreeWorkspace(ctx, col_flags);

  return {row_sorted, col_sorted};
}

template std::pair<bool, bool> COOIsSorted<kDLGPU, int32_t>(COOMatrix coo);
template std::pair<bool, bool> COOIsSorted<kDLGPU, int64_t>(COOMatrix coo);

}  // namespace impl
}  // namespace aten
}  // namespace dgl

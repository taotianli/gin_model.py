#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Contributors
 * \file array/cuda/array_scatter.cu
 * \brief Array scatter GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename DType, typename IdType>
__global__ void _ScatterKernel(const IdType* index, const DType* value,
                               int64_t length, DType* out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    out[index[tx]] = value[tx];
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename DType, typename IdType>
void Scatter_(IdArray index, NDArray value, NDArray out) {
  const int64_t len = index->shape[0];
  const IdType* idx = index.Ptr<IdType>();
  const DType* val = value.Ptr<DType>();
  DType* outd = out.Ptr<DType>();

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const int nt = cuda::FindNumThreads(len);
  const int nb = (len + nt - 1) / nt;
  CUDA_KERNEL_CALL(_ScatterKernel, nb, nt, 0, thr_entry->stream,
      idx, val, len, outd);
}

template void Scatter_<kDLGPU, int32_t, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, int64_t, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, float, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, double, int32_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, int32_t, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, int64_t, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, float, int64_t>(IdArray, NDArray, NDArray);
template void Scatter_<kDLGPU, double, int64_t>(IdArray, NDArray, NDArray);

};  // namespace impl
};  // namespace aten
};  // namespace dgl

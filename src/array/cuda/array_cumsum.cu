/*!
 *  Copyright (c) 2020 by Contributors
 * \file array/cpu/array_cumsum.cu
 * \brief Array cumsum GPU implementation
 */
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"
#include "./dgl_cub.cuh"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <DLDeviceType XPU, typename IdType>
IdArray CumSum(IdArray array, bool prepend_zero) {
  const int64_t len = array.NumElements();
  if (len == 0)
    return !prepend_zero ? array : aten::Full(0, 1, array->dtype.bits, array->ctx);

  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(array->ctx);
  const IdType* in_d = array.Ptr<IdType>();
  IdArray ret;
  IdType* out_d = nullptr;
  if (prepend_zero) {
    ret = aten::Full(0, len + 1, array->dtype.bits, array->ctx);
    out_d = ret.Ptr<IdType>() + 1;
  } else {
    ret = aten::NewIdArray(len, array->ctx, array->dtype.bits);
    out_d = ret.Ptr<IdType>();
  }
  // Allocate workspace
  size_t workspace_size = 0;
  CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
      nullptr, workspace_size, in_d, out_d, len, thr_entry->stream));
  void* workspace = device->AllocWorkspace(array->ctx, workspace_size);

  // Compute cumsum
  CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
      workspace, workspace_size, in_d, out_d, len, thr_entry->stream));

  device->FreeWorkspace(array->ctx, workspace);

  return ret;
}

template IdArray CumSum<kDLGPU, int32_t>(IdArray, bool);
template IdArray CumSum<kDLGPU, int64_t>(IdArray, bool);

}  // namespace impl
}  // namespace aten
}  // namespace dgl

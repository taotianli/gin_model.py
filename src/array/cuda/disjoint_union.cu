#include "hip/hip_runtime.h"
/**
*   Copyright (c) 2022, NVIDIA CORPORATION.
*
*   Licensed under the Apache License, Version 2.0 (the "License");
*   you may not use this file except in compliance with the License.
*   You may obtain a copy of the License at
*
*       http://www.apache.org/licenses/LICENSE-2.0
*
*   Unless required by applicable law or agreed to in writing, software
*   distributed under the License is distributed on an "AS IS" BASIS,
*   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
*   See the License for the specific language governing permissions and
*   limitations under the License.
*
* \file array/gpu/disjoint_union.cu
* \brief Disjoint union GPU implementation.
*/

#include <dgl/runtime/parallel_for.h>
#include <dgl/array.h>
#include <vector>
#include <tuple>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"

namespace dgl {
using runtime::NDArray;
namespace aten {
namespace impl {

template <typename IdType>
__global__ void _DisjointUnionKernel(
    IdType** arrs, IdType* prefix, IdType* offset, IdType* out,
    int64_t n_arrs, int n_elms) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < n_elms) {
    IdType i = dgl::cuda::_UpperBound(offset, n_arrs, tx) - 1;
    if (arrs[i] == NULL) {
      out[tx] = tx;
    } else {
      IdType j = tx - offset[i];
      out[tx] = arrs[i][j] + prefix[i];
    }
    tx += stride_x;
  }
}

template <DLDeviceType XPU, typename IdType>
std::tuple<IdArray, IdArray, IdArray> _ComputePrefixSums(const std::vector<COOMatrix>& coos) {
  IdType n = coos.size(), nbits = coos[0].row->dtype.bits;
  IdArray n_rows = NewIdArray(n, CPU, nbits);
  IdArray n_cols = NewIdArray(n, CPU, nbits);
  IdArray n_elms = NewIdArray(n, CPU, nbits);

  IdType* n_rows_data = n_rows.Ptr<IdType>();
  IdType* n_cols_data = n_cols.Ptr<IdType>();
  IdType* n_elms_data = n_elms.Ptr<IdType>();

  dgl::runtime::parallel_for(0, coos.size(), [&](IdType b, IdType e){
    for (IdType i = b; i < e; ++i) {
      n_rows_data[i] = coos[i].num_rows;
      n_cols_data[i] = coos[i].num_cols;
      n_elms_data[i] = coos[i].row->shape[0];
    }
  });

  return std::make_tuple(CumSum(n_rows.CopyTo(coos[0].row->ctx), true),
                         CumSum(n_cols.CopyTo(coos[0].row->ctx), true),
                         CumSum(n_elms.CopyTo(coos[0].row->ctx), true));
}

template <DLDeviceType XPU, typename IdType>
void _Merge(IdType** arrs, IdType* prefix, IdType* offset, IdType* out,
            int64_t n_arrs, int n_elms,
            DGLContext ctx, DGLType dtype, hipStream_t stream) {
  auto device = runtime::DeviceAPI::Get(ctx);
  int nt = 256;
  int nb = (n_elms + nt - 1) / nt;

  IdType** arrs_dev = static_cast<IdType**>(
      device->AllocWorkspace(ctx, n_arrs*sizeof(IdType*)));

  device->CopyDataFromTo(
      arrs, 0, arrs_dev, 0, sizeof(IdType*)*n_arrs,
      DGLContext{kDLCPU, 0}, ctx, dtype, 0);

  CUDA_KERNEL_CALL(_DisjointUnionKernel,
      nb, nt, 0, stream,
      arrs_dev, prefix, offset,
      out, n_arrs, n_elms);

  device->FreeWorkspace(ctx, arrs_dev);
}

template <DLDeviceType XPU, typename IdType>
COOMatrix DisjointUnionCoo(const std::vector<COOMatrix>& coos) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  auto device = runtime::DeviceAPI::Get(coos[0].row->ctx);
  uint64_t src_offset = 0, dst_offset = 0;
  bool has_data = false;
  bool row_sorted = true;
  bool col_sorted = true;

  // check if data index array
  for (size_t i = 0; i < coos.size(); ++i) {
    CHECK_SAME_DTYPE(coos[0].row, coos[i].row);
    CHECK_SAME_CONTEXT(coos[0].row, coos[i].row);
    has_data |= COOHasData(coos[i]);
  }

  auto prefixes = _ComputePrefixSums<XPU, IdType>(coos);
  auto prefix_src = static_cast<IdType*>(std::get<0>(prefixes)->data);
  auto prefix_dst = static_cast<IdType*>(std::get<1>(prefixes)->data);
  auto prefix_elm = static_cast<IdType*>(std::get<2>(prefixes)->data);

  std::unique_ptr<IdType*[]> rows(new IdType*[coos.size()]);
  std::unique_ptr<IdType*[]> cols(new IdType*[coos.size()]);
  std::unique_ptr<IdType*[]> data(new IdType*[coos.size()]);

  for (size_t i = 0; i < coos.size(); i++) {
    row_sorted &= coos[i].row_sorted;
    col_sorted &= coos[i].col_sorted;
    rows[i] = coos[i].row.Ptr<IdType>();
    cols[i] = coos[i].col.Ptr<IdType>();
    data[i] = coos[i].data.Ptr<IdType>();
  }

  auto ctx = coos[0].row->ctx;
  auto dtype = coos[0].row->dtype;
  auto stream = thr_entry->stream;

  IdType n_elements = 0;
  device->CopyDataFromTo(
      &prefix_elm[coos.size()], 0, &n_elements, 0,
      sizeof(IdType), coos[0].row->ctx, DGLContext{kDLCPU, 0},
      coos[0].row->dtype, 0);

  device->CopyDataFromTo(
      &prefix_src[coos.size()], 0, &src_offset, 0,
      sizeof(IdType), coos[0].row->ctx, DGLContext{kDLCPU, 0},
      coos[0].row->dtype, 0);

  device->CopyDataFromTo(
      &prefix_dst[coos.size()], 0, &dst_offset, 0,
      sizeof(IdType), coos[0].row->ctx, DGLContext{kDLCPU, 0},
      coos[0].row->dtype, 0);

  // Union src array
  IdArray result_src = NewIdArray(
    n_elements, coos[0].row->ctx, coos[0].row->dtype.bits);
  _Merge<XPU, IdType>(rows.get(), prefix_src, prefix_elm, result_src.Ptr<IdType>(),
         coos.size(), n_elements, ctx, dtype, stream);

  // Union dst array
  IdArray result_dst = NewIdArray(
    n_elements, coos[0].col->ctx, coos[0].col->dtype.bits);
  _Merge<XPU, IdType>(cols.get(), prefix_dst, prefix_elm, result_dst.Ptr<IdType>(),
         coos.size(), n_elements, ctx, dtype, stream);

  // Union data array if exists and fetch number of elements
  IdArray result_dat = NullArray();
  if (has_data) {
    result_dat =  NewIdArray(
      n_elements, coos[0].row->ctx, coos[0].row->dtype.bits);
    _Merge<XPU, IdType>(data.get(), prefix_elm, prefix_elm, result_dat.Ptr<IdType>(),
          coos.size(), n_elements, ctx, dtype, stream);
  }

  return COOMatrix(
    src_offset, dst_offset,
    result_src,
    result_dst,
    result_dat,
    row_sorted,
    col_sorted);
}

template COOMatrix DisjointUnionCoo<kDLGPU, int32_t>(const std::vector<COOMatrix>& coos);
template COOMatrix DisjointUnionCoo<kDLGPU, int64_t>(const std::vector<COOMatrix>& coos);

}  // namespace impl
}  // namespace aten
}  // namespace dgl

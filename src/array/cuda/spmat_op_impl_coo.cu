#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by contributors.
 * \file array/cuda/spmat_op_impl_coo.cu
 * \brief COO operator GPU implementation
 */
#include <dgl/array.h>
#include <vector>
#include <unordered_set>
#include <numeric>
#include "../../runtime/cuda/cuda_common.h"
#include "./utils.h"
#include "./atomic.cuh"

namespace dgl {

using runtime::NDArray;
using namespace cuda;

namespace aten {
namespace impl {


template <typename IdType>
__device__ void _warpReduce(volatile IdType *sdata, IdType tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

template <typename IdType>
__global__ void _COOGetRowNNZKernel(
    const IdType* __restrict__ row_indices,
    IdType* __restrict__ glb_cnt,
    const int64_t row_query,
    IdType nnz) {
  __shared__ IdType local_cnt[1024];
  IdType tx = threadIdx.x;
  IdType bx = blockIdx.x;
  local_cnt[tx] = 0;
  IdType start = bx * blockDim.x;
  while (start < nnz) {
    if (start + tx < nnz)
      local_cnt[tx] = (row_indices[start + tx] == row_query);
    __syncthreads();
    if (tx < 512) {
      local_cnt[tx] += local_cnt[tx + 512];
      __syncthreads();
    }
    if (tx < 256) {
      local_cnt[tx] += local_cnt[tx + 256];
      __syncthreads();
    }
    if (tx < 128) {
      local_cnt[tx] += local_cnt[tx + 128];
      __syncthreads();
    }
    if (tx < 64) {
      local_cnt[tx] += local_cnt[tx + 64];
      __syncthreads();
    }
    if (tx < 32) {
      _warpReduce(local_cnt, tx);
    }
    if (tx == 0) {
      cuda::AtomicAdd(glb_cnt, local_cnt[tx]);
    }
    start += blockDim.x * gridDim.x;
  }
}

template <DLDeviceType XPU, typename IdType>
int64_t COOGetRowNNZ(COOMatrix coo, int64_t row) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const auto& ctx = coo.row->ctx;
  IdType nnz = coo.row->shape[0];
  IdType nt = 1024;
  IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
  NDArray rst = NDArray::Empty({1}, coo.row->dtype, coo.row->ctx);
  _Fill(rst.Ptr<IdType>(), 1, IdType(0));
  CUDA_KERNEL_CALL(_COOGetRowNNZKernel,
      nb, nt, 0, thr_entry->stream,
      coo.row.Ptr<IdType>(), rst.Ptr<IdType>(),
      row, nnz);
  rst = rst.CopyTo(DLContext{kDLCPU, 0});
  return *rst.Ptr<IdType>();
}

template int64_t COOGetRowNNZ<kDLGPU, int32_t>(COOMatrix, int64_t);
template int64_t COOGetRowNNZ<kDLGPU, int64_t>(COOMatrix, int64_t);

template <typename IdType>
__global__ void _COOGetAllRowNNZKernel(
    const IdType* __restrict__ row_indices,
    IdType* __restrict__ glb_cnts,
    IdType nnz) {
  IdType eid = blockIdx.x * blockDim.x + threadIdx.x;
  while (eid < nnz) {
    IdType row = row_indices[eid];
    cuda::AtomicAdd(glb_cnts + row, IdType(1));
    eid += blockDim.x * gridDim.x;
  }
}

template <DLDeviceType XPU, typename IdType>
NDArray COOGetRowNNZ(COOMatrix coo, NDArray rows) {
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  const auto& ctx = coo.row->ctx;
  IdType nnz = coo.row->shape[0];
  IdType num_rows = coo.num_rows;
  IdType num_queries = rows->shape[0];
  if (num_queries == 1) {
    auto rows_cpu = rows.CopyTo(DLContext{kDLCPU, 0});
    int64_t row = *rows_cpu.Ptr<IdType>();
    IdType nt = 1024;
    IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
    NDArray rst = NDArray::Empty({1}, coo.row->dtype, coo.row->ctx);
    _Fill(rst.Ptr<IdType>(), 1, IdType(0));
    CUDA_KERNEL_CALL(_COOGetRowNNZKernel,
        nb, nt, 0, thr_entry->stream,
        coo.row.Ptr<IdType>(), rst.Ptr<IdType>(),
        row, nnz);
    return rst;
  } else {
    IdType nt = 1024;
    IdType nb = dgl::cuda::FindNumBlocks<'x'>((nnz + nt - 1) / nt);
    NDArray in_degrees = NDArray::Empty({num_rows}, rows->dtype, rows->ctx);
    _Fill(in_degrees.Ptr<IdType>(), num_rows, IdType(0));
    CUDA_KERNEL_CALL(_COOGetAllRowNNZKernel,
        nb, nt, 0, thr_entry->stream,
        coo.row.Ptr<IdType>(), in_degrees.Ptr<IdType>(),
        nnz);
    return IndexSelect(in_degrees, rows);
  }
}

template NDArray COOGetRowNNZ<kDLGPU, int32_t>(COOMatrix, NDArray);
template NDArray COOGetRowNNZ<kDLGPU, int64_t>(COOMatrix, NDArray);

}  // namespace impl
}  // namespace aten
}  // namespace dgl

#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file array/cuda/cuda_filter.cc
 * \brief Object for selecting items in a set, or selecting items not in a set.
 */

#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "../filter.h"
#include "../../runtime/cuda/cuda_hashtable.cuh"
#include "./dgl_cub.cuh"

using namespace dgl::runtime::cuda;

namespace dgl {
namespace array {

namespace {

// TODO(nv-dlasalle): Replace with getting the stream from the context
// when it's implemented.
constexpr hipStream_t cudaDefaultStream = 0;

template<typename IdType, bool include>
__global__ void _IsInKernel(
    DeviceOrderedHashTable<IdType> table,
    const IdType * const array,
    const int64_t size,
    IdType * const mark) {
  const int64_t idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < size) {
    mark[idx] = table.Contains(array[idx]) ^ (!include);
  }
}

template<typename IdType>
__global__ void _InsertKernel(
    const IdType * const prefix,
    const int64_t size,
    IdType * const result) {
  const int64_t idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx < size) {
    if (prefix[idx] != prefix[idx+1]) {
      result[prefix[idx]] = idx;
    }
  }
}

template<typename IdType, bool include>
IdArray _PerformFilter(
    const OrderedHashTable<IdType>& table,
    IdArray test) {
  const auto& ctx = test->ctx;
  auto device = runtime::DeviceAPI::Get(ctx);
  const int64_t size = test->shape[0];

  if (size == 0) {
    return test;
  }

  hipStream_t stream = cudaDefaultStream;

  // we need two arrays: 1) to act as a prefixsum
  // for the number of entries that will be inserted, and
  // 2) to collect the included items.
  IdType * prefix = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*(size+1)));

  // will resize down later
  IdArray result = aten::NewIdArray(size, ctx, sizeof(IdType)*8);

  // mark each index based on it's existence in the hashtable
  {
    const dim3 block(256);
    const dim3 grid((size+block.x-1)/block.x);

    CUDA_KERNEL_CALL((_IsInKernel<IdType, include>),
        grid, block, 0, stream,
        table.DeviceHandle(),
        static_cast<const IdType*>(test->data),
        size,
        prefix);
  }

  // generate prefix-sum
  {
    size_t workspace_bytes;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        nullptr,
        workspace_bytes,
        static_cast<IdType*>(nullptr),
        static_cast<IdType*>(nullptr),
        size+1));
    void * workspace = device->AllocWorkspace(ctx, workspace_bytes);

    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
        workspace,
        workspace_bytes,
        prefix,
        prefix,
        size+1, stream));
    device->FreeWorkspace(ctx, workspace);
  }

  // copy number
  IdType num_unique;
  device->CopyDataFromTo(prefix+size, 0,
      &num_unique, 0,
      sizeof(num_unique),
      ctx,
      DGLContext{kDLCPU, 0},
      test->dtype,
      stream);

  // insert items into set
  {
    const dim3 block(256);
    const dim3 grid((size+block.x-1)/block.x);

    CUDA_KERNEL_CALL(_InsertKernel,
        grid, block, 0, stream,
        prefix,
        size,
        static_cast<IdType*>(result->data));
  }
  device->FreeWorkspace(ctx, prefix);

  return result.CreateView({num_unique}, result->dtype);
}


template<typename IdType>
class CudaFilterSet : public Filter {
 public:
  explicit CudaFilterSet(IdArray array) :
      table_(array->shape[0], array->ctx, cudaDefaultStream) {
    table_.FillWithUnique(
        static_cast<const IdType*>(array->data),
        array->shape[0],
        cudaDefaultStream);
  }

  IdArray find_included_indices(IdArray test) override {
    return _PerformFilter<IdType, true>(table_, test);
  }

  IdArray find_excluded_indices(IdArray test) override {
    return _PerformFilter<IdType, false>(table_, test);
  }

 private:
  OrderedHashTable<IdType> table_;
};

}  // namespace

template<DLDeviceType XPU, typename IdType>
FilterRef CreateSetFilter(IdArray set) {
  return FilterRef(std::make_shared<CudaFilterSet<IdType>>(set));
}

template FilterRef CreateSetFilter<kDLGPU, int32_t>(IdArray set);
template FilterRef CreateSetFilter<kDLGPU, int64_t>(IdArray set);

}  // namespace array
}  // namespace dgl

#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file graph/sampling/randomwalk_gpu.cu
 * \brief DGL sampler 
 */

#include <dgl/array.h>
#include <dgl/base_heterograph.h>
#include <dgl/runtime/device_api.h>
#include <dgl/random.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <utility>
#include <tuple>

#include "../../../runtime/cuda/cuda_common.h"
#include "frequency_hashmap.cuh"

namespace dgl {

using namespace dgl::runtime;
using namespace dgl::aten;

namespace sampling {

namespace impl {

namespace {

template<typename IdType>
struct GraphKernelData {
  const IdType *in_ptr;
  const IdType *in_cols;
  const IdType *data;
};

template<typename IdType, typename FloatType, int BLOCK_SIZE, int TILE_SIZE>
__global__ void _RandomWalkKernel(
    const uint64_t rand_seed, const IdType *seed_data, const int64_t num_seeds,
    const IdType* metapath_data, const uint64_t max_num_steps,
    const GraphKernelData<IdType>* graphs,
    const FloatType* restart_prob_data,
    const int64_t restart_prob_size,
    const int64_t max_nodes,
    IdType *out_traces_data,
    IdType *out_eids_data) {
  assert(BLOCK_SIZE == blockDim.x);
  int64_t idx = blockIdx.x * TILE_SIZE + threadIdx.x;
  int64_t last_idx = min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_seeds);
  int64_t trace_length = (max_num_steps + 1);
  hiprandState rng;
  // reference:
  //     https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#performance-notes
  hiprand_init(rand_seed + idx, 0, 0, &rng);

  while (idx < last_idx) {
    IdType curr = seed_data[idx];
    assert(curr < max_nodes);
    IdType *traces_data_ptr = &out_traces_data[idx * trace_length];
    IdType *eids_data_ptr = &out_eids_data[idx * max_num_steps];
    *(traces_data_ptr++) = curr;
    int64_t step_idx;
    for (step_idx = 0; step_idx < max_num_steps; ++step_idx) {
      IdType metapath_id = metapath_data[step_idx];
      const GraphKernelData<IdType> &graph = graphs[metapath_id];
      const int64_t in_row_start = graph.in_ptr[curr];
      const int64_t deg = graph.in_ptr[curr + 1] - graph.in_ptr[curr];
      if (deg == 0) {  // the degree is zero
        break;
      }
      const int64_t num = hiprand(&rng) % deg;
      IdType pick = graph.in_cols[in_row_start + num];
      IdType eid = (graph.data? graph.data[in_row_start + num] : in_row_start + num);
      *traces_data_ptr = pick;
      *eids_data_ptr = eid;
      if ((restart_prob_size > 1) && (hiprand_uniform(&rng) < restart_prob_data[step_idx])) {
        break;
      } else if ((restart_prob_size == 1) && (hiprand_uniform(&rng) < restart_prob_data[0])) {
        break;
      }
      ++traces_data_ptr; ++eids_data_ptr;
      curr = pick;
    }
    for (; step_idx < max_num_steps; ++step_idx) {
      *(traces_data_ptr++) = -1;
      *(eids_data_ptr++) = -1;
    }
    idx += BLOCK_SIZE;
  }
}

}  // namespace

// random walk for uniform choice
template<DLDeviceType XPU, typename IdType>
std::pair<IdArray, IdArray> RandomWalkUniform(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    FloatArray restart_prob) {
  const int64_t max_num_steps = metapath->shape[0];
  const IdType *metapath_data = static_cast<IdType *>(metapath->data);
  const int64_t begin_ntype = hg->meta_graph()->FindEdge(metapath_data[0]).first;
  const int64_t max_nodes = hg->NumVertices(begin_ntype);
  int64_t num_etypes = hg->NumEdgeTypes();
  auto ctx = seeds->ctx;

  const IdType *seed_data = static_cast<const IdType*>(seeds->data);
  CHECK(seeds->ndim == 1) << "seeds shape is not one dimension.";
  const int64_t num_seeds = seeds->shape[0];
  int64_t trace_length = max_num_steps + 1;
  IdArray traces = IdArray::Empty({num_seeds, trace_length}, seeds->dtype, ctx);
  IdArray eids = IdArray::Empty({num_seeds, max_num_steps}, seeds->dtype, ctx);
  IdType *traces_data = traces.Ptr<IdType>();
  IdType *eids_data = eids.Ptr<IdType>();

  std::vector<GraphKernelData<IdType>> h_graphs(num_etypes);
  for (int64_t etype = 0; etype < num_etypes; ++etype) {
    const CSRMatrix &csr = hg->GetCSRMatrix(etype);
    h_graphs[etype].in_ptr  = static_cast<const IdType*>(csr.indptr->data);
    h_graphs[etype].in_cols = static_cast<const IdType*>(csr.indices->data);
    h_graphs[etype].data = (CSRHasData(csr) ? static_cast<const IdType*>(csr.data->data) : nullptr);
  }
  // use default stream
  hipStream_t stream = 0;
  auto device = DeviceAPI::Get(ctx);
  auto d_graphs = static_cast<GraphKernelData<IdType>*>(
      device->AllocWorkspace(ctx, (num_etypes) * sizeof(GraphKernelData<IdType>)));
  // copy graph metadata pointers to GPU
  device->CopyDataFromTo(h_graphs.data(), 0, d_graphs, 0,
      (num_etypes) * sizeof(GraphKernelData<IdType>),
      DGLContext{kDLCPU, 0},
      ctx,
      hg->GetCSRMatrix(0).indptr->dtype,
      stream);
  // copy metapath to GPU
  auto d_metapath = metapath.CopyTo(ctx);
  const IdType *d_metapath_data = static_cast<IdType *>(d_metapath->data);

  constexpr int BLOCK_SIZE = 256;
  constexpr int TILE_SIZE = BLOCK_SIZE * 4;
  dim3 block(256);
  dim3 grid((num_seeds + TILE_SIZE - 1) / TILE_SIZE);
  const uint64_t random_seed = RandomEngine::ThreadLocal()->RandInt(1000000000);
  ATEN_FLOAT_TYPE_SWITCH(restart_prob->dtype, FloatType, "random walk GPU kernel", {
    CHECK(restart_prob->ctx.device_type == kDLGPU) << "restart prob should be in GPU.";
    CHECK(restart_prob->ndim == 1) << "restart prob dimension should be 1.";
    const FloatType *restart_prob_data = restart_prob.Ptr<FloatType>();
    const int64_t restart_prob_size = restart_prob->shape[0];
    CUDA_KERNEL_CALL(
      (_RandomWalkKernel<IdType, FloatType, BLOCK_SIZE, TILE_SIZE>),
      grid, block, 0, stream,
      random_seed,
      seed_data,
      num_seeds,
      d_metapath_data,
      max_num_steps,
      d_graphs,
      restart_prob_data,
      restart_prob_size,
      max_nodes,
      traces_data,
      eids_data);
  });

  device->FreeWorkspace(ctx, d_graphs);
  return std::make_pair(traces, eids);
}

template<DLDeviceType XPU, typename IdType>
std::pair<IdArray, IdArray> RandomWalk(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob) {

  // not support no-uniform choice now
  for (const auto &etype_prob : prob) {
    if (!IsNullArray(etype_prob)) {
      LOG(FATAL) << "Non-uniform choice is not supported in GPU.";
    }
  }

  auto restart_prob = NDArray::Empty(
      {0}, DLDataType{kDLFloat, 32, 1}, DGLContext{XPU, 0});
  return RandomWalkUniform<XPU, IdType>(hg, seeds, metapath, restart_prob);
}

template<DLDeviceType XPU, typename IdType>
std::pair<IdArray, IdArray> RandomWalkWithRestart(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    double restart_prob) {

  // not support no-uniform choice now
  for (const auto &etype_prob : prob) {
    if (!IsNullArray(etype_prob)) {
      LOG(FATAL) << "Non-uniform choice is not supported in GPU.";
    }
  }
  auto device_ctx = seeds->ctx;
  auto restart_prob_array = NDArray::Empty(
      {1}, DLDataType{kDLFloat, 64, 1}, device_ctx);
  auto device = dgl::runtime::DeviceAPI::Get(device_ctx);

  // use default stream
  hipStream_t stream = 0;
  device->CopyDataFromTo(
      &restart_prob, 0, restart_prob_array.Ptr<double>(), 0,
      sizeof(double),
      DGLContext{kDLCPU, 0}, device_ctx,
      restart_prob_array->dtype, stream);
  device->StreamSync(device_ctx, stream);

  return RandomWalkUniform<XPU, IdType>(hg, seeds, metapath, restart_prob_array);
}

template<DLDeviceType XPU, typename IdType>
std::pair<IdArray, IdArray> RandomWalkWithStepwiseRestart(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    FloatArray restart_prob) {

  // not support no-uniform choice now
  for (const auto &etype_prob : prob) {
    if (!IsNullArray(etype_prob)) {
      LOG(FATAL) << "Non-uniform choice is not supported in GPU.";
    }
  }

  return RandomWalkUniform<XPU, IdType>(hg, seeds, metapath, restart_prob);
}

template<DLDeviceType XPU, typename IdxType>
std::tuple<IdArray, IdArray, IdArray> SelectPinSageNeighbors(
    const IdArray src,
    const IdArray dst,
    const int64_t num_samples_per_node,
    const int64_t k) {
  CHECK(src->ctx.device_type == kDLGPU) <<
    "IdArray needs be on GPU!";
  const IdxType* src_data = src.Ptr<IdxType>();
  const IdxType* dst_data = dst.Ptr<IdxType>();
  const int64_t num_dst_nodes = (dst->shape[0] / num_samples_per_node);
  auto ctx = src->ctx;
  // use default stream
  hipStream_t stream = 0;
  auto frequency_hashmap = FrequencyHashmap<IdxType>(num_dst_nodes,
      num_samples_per_node, ctx, stream);
  auto ret = frequency_hashmap.Topk(src_data, dst_data, src->dtype,
      src->shape[0], num_samples_per_node, k);
  return ret;
}

template
std::pair<IdArray, IdArray> RandomWalk<kDLGPU, int32_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob);
template
std::pair<IdArray, IdArray> RandomWalk<kDLGPU, int64_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob);

template
std::pair<IdArray, IdArray> RandomWalkWithRestart<kDLGPU, int32_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    double restart_prob);
template
std::pair<IdArray, IdArray> RandomWalkWithRestart<kDLGPU, int64_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    double restart_prob);

template
std::pair<IdArray, IdArray> RandomWalkWithStepwiseRestart<kDLGPU, int32_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    FloatArray restart_prob);
template
std::pair<IdArray, IdArray> RandomWalkWithStepwiseRestart<kDLGPU, int64_t>(
    const HeteroGraphPtr hg,
    const IdArray seeds,
    const TypeArray metapath,
    const std::vector<FloatArray> &prob,
    FloatArray restart_prob);

template
std::tuple<IdArray, IdArray, IdArray> SelectPinSageNeighbors<kDLGPU, int32_t>(
    const IdArray src,
    const IdArray dst,
    const int64_t num_samples_per_node,
    const int64_t k);
template
std::tuple<IdArray, IdArray, IdArray> SelectPinSageNeighbors<kDLGPU, int64_t>(
    const IdArray src,
    const IdArray dst,
    const int64_t num_samples_per_node,
    const int64_t k);


};  // namespace impl

};  // namespace sampling

};  // namespace dgl

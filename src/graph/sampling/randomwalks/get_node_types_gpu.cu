/*!
 *  Copyright (c) 2021 by Contributors
 * \file graph/sampling/get_node_types_gpu.cu
 * \brief DGL sampler
 */

#include <dgl/array.h>
#include <dgl/base_heterograph.h>
#include <dgl/runtime/device_api.h>
#include <hip/hip_runtime.h>
#include <utility>
#include "randomwalks_impl.h"

namespace dgl {

using namespace dgl::runtime;
using namespace dgl::aten;

namespace sampling {

namespace impl {

template<DLDeviceType XPU, typename IdxType>
TypeArray GetNodeTypesFromMetapath(
    const HeteroGraphPtr hg,
    const TypeArray metapath) {

  uint64_t num_etypes = metapath->shape[0];

  auto cpu_ctx = DGLContext{kDLCPU, 0};
  auto metapath_ctx = metapath->ctx;
  // use default stream
  hipStream_t stream = 0;

  TypeArray h_result = TypeArray::Empty(
      {metapath->shape[0] + 1}, metapath->dtype, cpu_ctx);
  auto h_result_data = h_result.Ptr<IdxType>();

  auto h_metapath = metapath.CopyTo(cpu_ctx, stream);
  DeviceAPI::Get(metapath_ctx)->StreamSync(metapath_ctx, stream);
  const IdxType *h_metapath_data = h_metapath.Ptr<IdxType>();

  dgl_type_t curr_type = hg->GetEndpointTypes(h_metapath_data[0]).first;
  h_result_data[0] = curr_type;

  for (uint64_t i = 0; i < num_etypes; ++i) {
    auto src_dst_type = hg->GetEndpointTypes(h_metapath_data[i]);
    dgl_type_t srctype = src_dst_type.first;
    dgl_type_t dsttype = src_dst_type.second;

    if (srctype != curr_type) {
      LOG(FATAL) << "source of edge type #" << i <<
        " does not match destination of edge type #" << i - 1;
    }
    curr_type = dsttype;
    h_result_data[i + 1] = dsttype;
  }

  auto result = h_result.CopyTo(metapath->ctx, stream);
  DeviceAPI::Get(metapath_ctx)->StreamSync(metapath_ctx, stream);
  return result;
}

template
TypeArray GetNodeTypesFromMetapath<kDLGPU, int32_t>(
    const HeteroGraphPtr hg,
    const TypeArray metapath);
template
TypeArray GetNodeTypesFromMetapath<kDLGPU, int64_t>(
    const HeteroGraphPtr hg,
    const TypeArray metapath);

};  // namespace impl

};  // namespace sampling

};  // namespace dgl

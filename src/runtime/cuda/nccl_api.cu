
#include <hip/hip_runtime.h>
/*!
 *  Copyright (c) 2021 by Contributors
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 * \file nccl_api.cu
 * \brief Implementation of wrapper around NCCL routines.
 */


#ifdef DGL_USE_NCCL

#include "nccl_api.h"

#include <dgl/array.h>
#include <dgl/aten/array_ops.h>
#include <dgl/runtime/container.h>
#include <dgl/runtime/device_api.h>
#include <dgl/packed_func_ext.h>
#include <dgl/runtime/registry.h>
#include <cuda_runtime.h>
#include <cuda_fp16.h>

#include <cmath>
#include <sstream>
#include <iomanip>
#include <utility>
#include <vector>
#include <memory>
#include <string>
#include <limits>

#include "cuda_common.h"
#include "../../runtime/workspace.h"
#include "../../partition/ndarray_partition.h"
#include "../../array/cuda/dgl_cub.cuh"
#include "../../array/cuda/array_index_select.cuh"

#define NCCL_CALL(func) \
{ \
  ncclResult_t result = func; \
  if (result != ncclSuccess) { \
      LOG(FATAL)                                                        \
          << "NCCLError: " #func " failed with error: " << result;            \
  } \
}

namespace dgl {

using namespace partition;

namespace runtime {
namespace cuda {

namespace {

enum class AllToAllMode : int {
  REMAINDER = 0
};


template<typename T> ncclDataType_t NCCLType();
template<> ncclDataType_t NCCLType<int32_t>() {
    return ncclInt32;
}
template<> ncclDataType_t NCCLType<int64_t>() {
    return ncclInt64;
}
template<> ncclDataType_t NCCLType<__half>() {
    return ncclHalf;
}
template<> ncclDataType_t NCCLType<float>() {
    return ncclFloat32;
}
template<> ncclDataType_t NCCLType<double>() {
    return ncclFloat64;
}


template<typename IdType, typename DType>
__global__ void _DualPermKernel(
    const IdType * const in_idx,
    const DType * const in_value,
    const IdType * const perm,
    const int64_t num_in,
    const int64_t num_feat,
    IdType * const out_idx,
    DType * const out_value) {
  // set index permutation
  const int64_t tidx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;
  if (tidx < num_in) {
    const IdType perm_idx = perm[tidx];
    assert(perm_idx < num_in);
    out_idx[tidx] = in_idx[perm_idx];
  }

  if (num_feat > 1) {
    for (int d = 0; d < blockDim.x; ++d) {
      const int64_t bidx = blockDim.x*static_cast<int64_t>(blockIdx.x) + d;
      if (bidx < num_in) {
        const IdType perm_idx = perm[bidx];
        for (int64_t f = threadIdx.x; f < num_feat; f+=blockDim.x) {
          out_value[bidx*num_feat+f] = in_value[perm_idx*num_feat+f];
        }
      }
    }
  } else {
    if (tidx < num_in) {
      const IdType perm_idx = perm[tidx];
      out_value[tidx] = in_value[perm_idx];
    }
  }
}

template <typename DType, typename IdType>
__global__ void _InversePermKernel(
        const DType* const array,
        const int64_t num_feat,
        int64_t length,
        const IdType* const perm,
        DType* const out) {
  int64_t in_row = blockIdx.x*blockDim.y+threadIdx.y;

  const int64_t stride = blockDim.y*gridDim.x;

  while (in_row < length) {
    int64_t col = threadIdx.x;
    const int64_t out_row = perm[in_row];
    while (col < num_feat) {
      out[out_row*num_feat+col] = array[in_row*num_feat+col];
      col += blockDim.x;
    }
    in_row += stride;
  }
}


template<typename IdType, typename DType>
std::pair<IdArray, NDArray> SparsePush(
    NCCLCommunicatorRef comm,
    IdArray in_idx,
    NDArray in_value,
    NDArrayPartitionRef part) {
  const auto& ctx = in_idx->ctx;
  CHECK_EQ(ctx, in_value->ctx) << "Indices and values must be on the same "
      "device";
  auto device = DeviceAPI::Get(ctx);

  // TODO(dlasalle): Get the stream from the device context.
  cudaStream_t stream = 0;

  CHECK_LE(in_idx->ndim, 1) << "The tensor of sending indices must be of "
      "dimension one (or empty).";
  const int64_t num_in = in_idx->ndim > 0 ? in_idx->shape[0] : 0;

  CHECK_EQ(num_in, in_value->ndim > 0 ? in_value->shape[0] : 0) <<
      "Leading dimension of indices (" << num_in << ") must match "
      "leading dimension of values (" <<
      (in_value->ndim > 0 ? in_value->shape[0] : 0) << ").";

  int64_t num_feat = 1;
  for (int d = 1; d < in_value->ndim; ++d) {
    num_feat *= in_value->shape[d];
  }

  const int64_t comm_size = comm->size();

  if (comm_size == 1) {
    // nothing to do, just return original arrays
    return std::pair<IdArray, NDArray>(in_idx, in_value);
  }

  std::pair<IdArray, NDArray> part_perm = part->GeneratePermutation(in_idx);
  const IdType * const perm = static_cast<const IdType*>(part_perm.first->data);
  const int64_t * const send_sum =
      static_cast<const int64_t*>(part_perm.second->data);

  Workspace<IdType> send_idx(device, ctx, num_in);
  Workspace<DType> send_value(device, ctx, num_in*num_feat);

  // permute the indices and values
  if (num_in > 0) {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    CUDA_KERNEL_CALL(_DualPermKernel,
        grid, block, 0, stream,
        static_cast<const IdType*>(in_idx->data),
        static_cast<const DType*>(in_value->data),
        perm,
        num_in,
        num_feat,
        send_idx.get(),
        send_value.get());
  }

  // compute the prefix sum of the send values
  Workspace<int64_t> send_prefix(device, ctx, comm_size+1);
  {
    size_t prefix_workspace_size;
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        send_sum, send_prefix.get(), comm_size+1, stream));

    Workspace<void> prefix_workspace(device, ctx, prefix_workspace_size);
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(prefix_workspace.get(),
        prefix_workspace_size, send_sum, send_prefix.get(),
        comm_size+1, stream));
  }

  std::vector<int64_t> send_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      send_prefix.get(),
      0,
      send_prefix_host.data(),
      0,
      send_prefix_host.size()*sizeof(*send_prefix.get()),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*send_prefix.get())*8, 1},
      stream);
  send_prefix.free();

  CHECK_EQ(send_prefix_host.back(), num_in) << "Internal Error: "
      "send_prefix_host.back() = " << send_prefix_host.back() <<
      ", and num_in = " << num_in;

  // communicate the amount to send
  Workspace<int64_t> recv_sum(device, ctx, comm_size+1);
  comm->AllToAll(send_sum, recv_sum.get(), 1, stream);

  cudaEvent_t d2h;
  CUDA_CALL(cudaEventCreate(&d2h));

  // compute the prefix sum of the recv values
  Workspace<int64_t> recv_prefix(device, ctx, comm_size+1);
  {
    size_t prefix_workspace_size;
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        recv_sum.get(), recv_prefix.get(), comm_size+1));

    Workspace<void> prefix_workspace(device, ctx, prefix_workspace_size);
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(prefix_workspace.get(),
        prefix_workspace_size, recv_sum.get(), recv_prefix.get(), comm_size+1));
  }
  recv_sum.free();

  // finally copy the prefixsum sum down to the host
  std::vector<int64_t> recv_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      recv_prefix.get(),
      0,
      recv_prefix_host.data(),
      0,
      recv_prefix_host.size()*sizeof(*recv_prefix.get()),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*recv_prefix.get())*8, 1},
      stream);
  recv_prefix.free();

  // use an event to track when copying is done
  CUDA_CALL(cudaEventRecord(d2h, stream));

  // allocate output space
  CUDA_CALL(cudaEventSynchronize(d2h));
  CUDA_CALL(cudaEventDestroy(d2h));

  IdArray recv_idx = aten::NewIdArray(
      recv_prefix_host.back(), ctx, sizeof(IdType)*8);

  std::vector<int64_t> value_shape(in_value->ndim, 0);
  value_shape[0] = recv_prefix_host.back();
  for (int d = 1; d < in_value->ndim; ++d) {
    value_shape[d] = in_value->shape[d];
  }
  NDArray recv_value = NDArray::Empty(value_shape, in_value->dtype, ctx);

  // send data
  comm->SparseAllToAll(
      send_idx.get(),
      send_value.get(),
      num_feat,
      send_prefix_host.data(),
      static_cast<IdType*>(recv_idx->data),
      static_cast<DType*>(recv_value->data),
      recv_prefix_host.data(),
      stream);

  return std::pair<IdArray, NDArray>(recv_idx, recv_value);
}

template<typename IdType, typename DType>
NDArray SparsePull(
    NCCLCommunicatorRef comm,
    IdArray req_idx,
    NDArray local_tensor,
    NDArrayPartitionRef part) {
  const auto& ctx = req_idx->ctx;
  CHECK_EQ(ctx, local_tensor->ctx) << "The request indices and set of local "
      "values must be on the same device";
  auto device = DeviceAPI::Get(ctx);

  cudaStream_t stream = CUDAThreadEntry::ThreadLocal()->stream;

  CHECK_LE(req_idx->ndim, 1) << "The tensor of requested indices must be of "
      "dimension one (or empty).";
  const int64_t num_in = req_idx->ndim > 0 ? req_idx->shape[0] : 0;
  int64_t num_feat = 1;
  for (int d = 1; d < local_tensor->ndim; ++d) {
    num_feat *= local_tensor->shape[d];
  }

  const int64_t comm_size = comm->size();

  if (comm_size == 1) {
    // Just return index selection from current local_tensor
    return aten::IndexSelect(local_tensor, req_idx);
  }

  // First we need to send our requests to other processors. This means
  // re-ordering our index array to be contiguous among processors, and
  // counting the number of indices we are sending each processor. For now,
  // we assume a poorly partitioned graph, and that there exists the
  // possibility that each processor could request data from this one.

  // the buffer for us to re-order our requests in
  Workspace<IdType> send_idx(device, ctx, num_in);

  std::pair<IdArray, NDArray> part_perm = part->GeneratePermutation(req_idx);
  const IdType * const perm = static_cast<const IdType*>(part_perm.first->data);
  const int64_t * const send_sum =
      static_cast<const int64_t*>(part_perm.second->data);

  // permute requests
  if (num_in > 0) {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    CUDA_KERNEL_CALL(aten::impl::IndexSelectSingleKernel,
        grid, block, 0, stream,
        static_cast<const IdType*>(req_idx->data),
        perm,
        num_in,
        req_idx->shape[0],
        send_idx.get());
  }

  // compute the prefix sum of the indexes this process is requesting
  Workspace<int64_t> request_prefix(device, ctx, comm_size+1);
  {
    size_t prefix_workspace_size;
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        send_sum, request_prefix.get(), comm_size+1, stream));

    Workspace<void> prefix_workspace(device, ctx, prefix_workspace_size);
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(prefix_workspace.get(),
        prefix_workspace_size, send_sum, request_prefix.get(),
        comm_size+1, stream));
  }

  cudaEvent_t d2h;
  CUDA_CALL(cudaEventCreate(&d2h));

  std::vector<int64_t> request_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      request_prefix.get(),
      0,
      request_prefix_host.data(),
      0,
      request_prefix_host.size()*sizeof(*request_prefix.get()),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*request_prefix.get())*8, 1},
      stream);
  request_prefix.free();
  CHECK_EQ(request_prefix_host.back(), num_in) << "Internal Error: "
      "request_prefix_host.back() = " << request_prefix_host.back() <<
      ", num_in = " << num_in;

  // communicate the amount requested
  Workspace<int64_t> recv_sum(device, ctx, comm_size+1);
  comm->AllToAll(send_sum, recv_sum.get(), 1, stream);

  // compute the prefix sum of the requested indexes
  Workspace<int64_t> response_prefix(device, ctx, comm_size+1);
  {
    size_t prefix_workspace_size;
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        recv_sum.get(), response_prefix.get(), comm_size+1, stream));

    Workspace<void> prefix_workspace(device, ctx, prefix_workspace_size);
    CUDA_CALL(cub::DeviceScan::ExclusiveSum(prefix_workspace.get(),
        prefix_workspace_size, recv_sum.get(), response_prefix.get(),
        comm_size+1, stream));
  }
  recv_sum.free();

  // finally copy the prefixsum sum down to the host
  std::vector<int64_t> response_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      response_prefix.get(),
      0,
      response_prefix_host.data(),
      0,
      response_prefix_host.size()*sizeof(*response_prefix.get()),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*response_prefix.get())*8, 1},
      stream);
  response_prefix.free();

  // use an event to track when copying is done
  CUDA_CALL(cudaEventRecord(d2h, stream));

  // allocate output space
  CUDA_CALL(cudaEventSynchronize(d2h));
  CUDA_CALL(cudaEventDestroy(d2h));

  // gather requested indexes
  IdArray recv_idx = aten::NewIdArray(
      response_prefix_host.back(), ctx, sizeof(IdType)*8);
  comm->AllToAllV(
      send_idx.get(),
      request_prefix_host.data(),
      static_cast<IdType*>(recv_idx->data),
      response_prefix_host.data(),
      stream);
  send_idx.free();

  // convert requested indices to local indices depending on partition
  if (response_prefix_host.back() > 0) {
    recv_idx = part->MapToLocal(recv_idx);
  }

  // and then index select them into place
  Workspace<DType> filled_response_value(device, ctx,
      response_prefix_host.back()*num_feat);
  if (response_prefix_host.back() > 0) {
    dim3 block(256, 1);
    while (block.x >= 2*num_feat) {
        block.x /= 2;
        block.y *= 2;
    }
    const dim3 grid((response_prefix_host.back()+block.y-1)/block.y);

    CUDA_KERNEL_CALL(aten::impl::IndexSelectMultiKernel,
        grid, block, 0, stream,
        static_cast<const DType*>(local_tensor->data),
        num_feat,
        static_cast<IdType*>(recv_idx->data),
        response_prefix_host.back(),
        local_tensor->shape[0],
        filled_response_value.get());
  }

  // we will collect recieved values in this array
  std::vector<int64_t> value_shape(local_tensor->ndim, 0);
  value_shape[0] = request_prefix_host.back();
  for (int d = 1; d < local_tensor->ndim; ++d) {
    value_shape[d] = local_tensor->shape[d];
  }
  Workspace<DType> filled_request_value(device, ctx,
      request_prefix_host.back()*num_feat);

  // multiply the prefixes by the number of features being sent
  for (auto& v : request_prefix_host) {
    v *= num_feat;
  }
  for (auto& v : response_prefix_host) {
    v *= num_feat;
  }

  // send the values
  comm->AllToAllV(
      filled_response_value.get(),
      response_prefix_host.data(),
      filled_request_value.get(),
      request_prefix_host.data(),
      stream);
  filled_response_value.free();

  // finally, we need to permute the values back into the requested order
  NDArray result = NDArray::Empty(value_shape, local_tensor->dtype, ctx);
  if (num_in > 0) {
    dim3 block(256, 1);
    while (block.x >= 2*num_feat) {
        block.x /= 2;
        block.y *= 2;
    }
    const dim3 grid((num_in+block.y-1)/block.y);

    CUDA_KERNEL_CALL(_InversePermKernel,
        grid, block, 0, stream,
        filled_request_value.get(),
        num_feat,
        num_in,
        perm,
        static_cast<DType*>(result->data));
  }

  return result;
}

}  // namespace

/* NCCLUniqueId **************************************************************/

NCCLUniqueId::NCCLUniqueId() :
  id_() {
  // this ID is unique to the process, not to each call of this function
  NCCL_CALL(ncclGetUniqueId(&id_));
}

ncclUniqueId NCCLUniqueId::Get() const {
  return id_;
}

std::string NCCLUniqueId::ToString() const {
  std::ostringstream oss;

  oss << std::hex;

  for (size_t b = 0; b < NCCL_UNIQUE_ID_BYTES; ++b) {
    const int num = static_cast<uint8_t>(id_.internal[b]);
    oss << std::setw(2) << std::setfill('0') << num;
  }

  std::string result = oss.str();
  CHECK_EQ(result.length(), NCCL_UNIQUE_ID_BYTES*2) <<
    "Invalid NCCL ID format: '" << result << "'";

  return result;
}

void NCCLUniqueId::FromString(
    const std::string& str) {
  // must be exactly 256 hex characters
  CHECK_EQ(str.length(), NCCL_UNIQUE_ID_BYTES * 2) <<
        "Invalid NCCL ID format: '" << str << "'";

  for (size_t b = 0; b < NCCL_UNIQUE_ID_BYTES; ++b) {
    id_.internal[b] = std::strtol(str.substr(b*2, 2).c_str(), nullptr, 16);
  }
}



/* NCCLCommunicator **********************************************************/

NCCLCommunicator::NCCLCommunicator(
    const int size,
    const int rank,
    ncclUniqueId id) :
  comm_(),
  size_(size),
  rank_(rank) {
  CHECK_LT(rank, size) << "The rank (" << rank << ") must be smaller than "
      "the size of the communicator (" << size << ").";
  CHECK_GE(rank, 0) << "The rank (" << rank << ") must be greater than or "
      "equal to 0.";

  NCCL_CALL(ncclCommInitRank(&comm_, size_, id, rank_));
}

NCCLCommunicator::~NCCLCommunicator() {
  ncclCommDestroy(comm_);
}

ncclComm_t NCCLCommunicator::Get() {
  return comm_;
}

template<typename DType>
void NCCLCommunicator::AllToAllV(
    const DType * const send,
    const int64_t * const send_prefix,
    DType * const recv,
    const int64_t * const recv_prefix,
    cudaStream_t stream) {
  const ncclDataType_t type = NCCLType<DType>();

  NCCL_CALL(ncclGroupStart());
  for (int r = 0; r < size_; ++r) {
    const int64_t send_size = send_prefix[r+1]-send_prefix[r];
    if (send_size > 0) {
      NCCL_CALL(ncclSend(send+send_prefix[r], send_size, type, r, comm_, stream));
    }
    const int64_t recv_size = recv_prefix[r+1]-recv_prefix[r];
    if (recv_size > 0) {
      NCCL_CALL(ncclRecv(recv+recv_prefix[r], recv_size, type, r, comm_, stream));
    }
  }
  NCCL_CALL(ncclGroupEnd());
}

template
void NCCLCommunicator::AllToAllV<int32_t>(
    const int32_t * const send,
    const int64_t * send_prefix,
    int32_t * const recv,
    const int64_t * recv_prefix,
    cudaStream_t stream);
template
void NCCLCommunicator::AllToAllV<int64_t>(
    const int64_t * const send,
    const int64_t * send_prefix,
    int64_t * const recv,
    const int64_t * recv_prefix,
    cudaStream_t stream);
template
void NCCLCommunicator::AllToAllV<float>(
    const float * const send,
    const int64_t * send_prefix,
    float * const recv,
    const int64_t * recv_prefix,
    cudaStream_t stream);
template
void NCCLCommunicator::AllToAllV<__half>(
    const __half * const send,
    const int64_t * send_prefix,
    __half * const recv,
    const int64_t * recv_prefix,
    cudaStream_t stream);


template<typename IdType>
void NCCLCommunicator::AllToAll(
    const IdType * const send,
    IdType * const recv,
    const int64_t count,
    cudaStream_t stream) {
  const ncclDataType_t type = NCCLType<IdType>();

  NCCL_CALL(ncclGroupStart());
  for (int r = 0; r < size_; ++r) {
    NCCL_CALL(ncclSend(send+(r*count), count, type, r, comm_, stream));
    NCCL_CALL(ncclRecv(recv+(r*count), count, type, r, comm_, stream));
  }
  NCCL_CALL(ncclGroupEnd());
}

template
void NCCLCommunicator::AllToAll<int32_t>(
    const int32_t * const send,
    int32_t * const recv,
    const int64_t count,
    cudaStream_t stream);
template
void NCCLCommunicator::AllToAll<int64_t>(
    const int64_t * const send,
    int64_t * const recv,
    const int64_t count,
    cudaStream_t stream);


template<typename IdType, typename DType>
void NCCLCommunicator::SparseAllToAll(
      const IdType * const send_idx,
      const DType * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      IdType * const recv_idx,
      DType * const recv_value,
      const int64_t * const recv_prefix,
      cudaStream_t stream) {
  const ncclDataType_t idx_type = NCCLType<IdType>();
  const ncclDataType_t value_type = NCCLType<DType>();

  // idxs
  AllToAllV(send_idx, send_prefix, recv_idx, recv_prefix, stream);

  // values
  NCCL_CALL(ncclGroupStart());
  for (int r = 0; r < size_; ++r) {
    const int64_t send_size = send_prefix[r+1]-send_prefix[r];
    if (send_size > 0) {
      NCCL_CALL(ncclSend(send_value+send_prefix[r]*num_feat, send_size*num_feat,
                         value_type, r, comm_, stream));
    }
    const int64_t recv_size = recv_prefix[r+1]-recv_prefix[r];
    if (recv_size > 0) {
      NCCL_CALL(ncclRecv(recv_value+recv_prefix[r]*num_feat, recv_size*num_feat,
                         value_type, r, comm_, stream));
    }
  }
  NCCL_CALL(ncclGroupEnd());
}


template
void NCCLCommunicator::SparseAllToAll<int32_t, __half>(
      const int32_t * const send_idx,
      const __half * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      int32_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      cudaStream_t stream);
template
void NCCLCommunicator::SparseAllToAll<int64_t, __half>(
      const int64_t * const send_idx,
      const __half * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      int64_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      cudaStream_t stream);

int NCCLCommunicator::size() const {
  return size_;
}

int NCCLCommunicator::rank() const {
  return rank_;
}


/* CAPI **********************************************************************/

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLGetUniqueId")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  *rv = NCCLUniqueIdRef(std::make_shared<NCCLUniqueId>());
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLUniqueIdToString")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLUniqueIdRef idObj = args[0];
  *rv = idObj->ToString();
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLUniqueIdFromString")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  const std::string str = args[0];

  NCCLUniqueIdRef ref(std::make_shared<NCCLUniqueId>());
  ref->FromString(str);
  *rv = ref;
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLCreateComm")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  const int size = args[0];
  const int rank = args[1];
  NCCLUniqueIdRef idObj = args[2];

  *rv = NCCLCommunicatorRef(std::make_shared<NCCLCommunicator>(size, rank,
        idObj->Get()));
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLSparseAllToAllPush")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLCommunicatorRef comm = args[0];
  IdArray in_idx = args[1];
  NDArray in_values = args[2];
  NDArrayPartitionRef part = args[3];

  List<ObjectRef> ret;
  ATEN_ID_TYPE_SWITCH(in_idx->dtype, IdType, {
    ATEN_DTYPE_SWITCH(in_values->dtype, DType, "values", {
      auto result = SparsePush<IdType, DType>(comm, in_idx, in_values, part);
      ret.push_back(Value(MakeValue(result.first)));
      ret.push_back(Value(MakeValue(result.second)));
    });
  });

  *rv = ret;
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLSparseAllToAllPull")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLCommunicatorRef comm = args[0];
  // the indexes this process is requesting from others
  IdArray req_idx = args[1];

  // the tensor this process has to fulfill other requests
  NDArray tensor = args[2];
  NDArrayPartitionRef part = args[3];

  ATEN_ID_TYPE_SWITCH(req_idx->dtype, IdType, {
    ATEN_DTYPE_SWITCH(tensor->dtype, DType, "values", {
      *rv = SparsePull<IdType, DType>(comm, req_idx, tensor, part);
    });
  });
});


}  // namespace cuda
}  // namespace runtime
}  // namespace dgl

#endif

